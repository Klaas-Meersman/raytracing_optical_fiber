#include "hip/hip_runtime.h"
#include "ray.hpp"
#include "fiber.hpp"
#include <cmath>
#include <numbers>

// Default constructor
__host__ __device__
Ray::Ray()
    : start(0, 0), end(0, 0), angleOfDeparture(0), fiber(nullptr), direction(Direction::UP), endHitFiber(false) {
}

// Constructor with Fiber pointer
__host__ __device__
Ray::Ray(Coordinate start, double_t angleOfDeparture, const Fiber* fiber)
    : start(start), angleOfDeparture(angleOfDeparture), fiber(fiber), direction(Direction::UP), endHitFiber(false) {
    //if (!fiber) return;
    if (angleOfDeparture > 0 && angleOfDeparture < M_PI / 2) {
        direction = Direction::UP;
        this->end.y = fiber->getTopY();
        this->end.x = this->start.x + (fiber->getTopY() - this->start.y) / std::tan(this->angleOfDeparture);
    } else if (angleOfDeparture > 3 * M_PI / 4 && angleOfDeparture < 2 * M_PI) {
        direction = Direction::DOWN;
        this->end.y = fiber->getBottomY();
        this->end.x = this->start.x + (fiber->getBottomY() - this->start.y) / std::tan(this->angleOfDeparture); 
    } else {
        // On device, don't throw: just mark as invalid

    }
    if(this->end.x > fiber->getLength()){
        this->endHitFiber = true;
        this->end.x = fiber->getLength();
        this->end.y = std::tan(this->angleOfDeparture) * (fiber->getLength() - this->start.x) + this->start.y;
    }
}





// CUDA-compatible propagateRay (in-place, returns void)


#ifndef __CUDA_ARCH__
// Host-only methods (using STL or exceptions) can go here
#endif
